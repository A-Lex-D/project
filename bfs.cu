#include "hip/hip_runtime.h"
/*
*
*	Graph builder from:
*	sanfoundry.com/c-program-generate-random-undirected-graph-given-number-edges/
*
*
*	BFS Traversal from
*   http://www.thelearningpoint.net/computer-science/algorithms-graph-traversal---breadth-first-search-with-c-program-source-code
*
*	
*	Modifications done to source code to get them to work together
*
*
*	Compile with:  nvcc -o bfs bfs.cu 
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include "cuPrintf.cuh"
#include "cuPrintf.cu"

#define VERTICES 20
#define MAX_EDGES 2
#define MAX_THREADS_PER_BLOCK 256

typedef struct Queue
{
	int capacity;
	int size;
	int front;
	int rear;
	int *elements;
}Queue;

struct Node
{
	int start;
	int no_of_edges;
};

// Assertion to check for errors

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void
Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask, bool* g_graph_visited, bool *g_over )
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<VERTICES && g_updating_graph_mask[tid])
	{

		g_graph_mask[tid]=true;
		g_graph_visited[tid]=true;
		*g_over=true;
		//cuPrintf("visting branches in kernal2 %d,", tid);
		g_updating_graph_mask[tid]=false;
	}
}


__global__ void
Kernel( Node* g_graph, int *g_edge, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited ) 
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<VERTICES && g_graph_mask[tid])
	{
		g_graph_mask[tid]=false;
		//cuPrintf("now visiting%d, %d\n", tid, g_graph[tid].start);
		for(int i=g_graph[tid].start; i<(g_graph[tid].no_of_edges+g_graph[tid].start); i++)
			{
			int id = g_edge[i];
			if(!g_graph_visited[id])
				{
			//cuPrintf("branches %d,", id);
				g_updating_graph_mask[id]=true;
				}
			}
	//cuPrintf("\n");
	}
}
void Bfs(Node* graph_nodes, int *graph_edge, int presentVertex,bool *visited);
int main()
{

	// GPU Timing variables
	hipEvent_t start, stop, start1, stop1;
	float elapsed_gpu, elapsed_gpu1;
	//int i;
	

	printf("Total vertices = %d, Max edges = %d\n", VERTICES, MAX_EDGES);

	Node* graph_nodes = (Node*) malloc(sizeof(Node)*VERTICES);
	
	int* graph_edge; //= (int *) malloc(sizeof(int)*VERTICES*MAX_EDGES);
	
	//Generate Varibles to check
	bool *graph_mask;
	if ((graph_mask = (bool*) malloc(sizeof(bool)*VERTICES))==NULL)
	{
		printf("Could not allocate memory for graph_mask\n");
		exit(1);
	}

	bool *updating_graph_mask;
	if ((updating_graph_mask = (bool*) malloc(sizeof(bool)*VERTICES))==NULL)
	{
		printf("Could not allocate memory for graph_mask\n");
		exit(1);
	}
	bool *graph_visited;
	if ((graph_visited = (bool*) malloc(sizeof(bool)*VERTICES))==NULL)
	{
		printf("Could not allocate memory for graph_mask\n");
		exit(1);
	}
	bool *h_graph_visited;
	if ((h_graph_visited = (bool*) malloc(sizeof(bool)*VERTICES))==NULL)
	{
		printf("Could not allocate memory for graph_mask\n");
		exit(1);
	}
	//for(i=0; i<VERTICES*MAX_EDGES; i++)
	//graph_edge[i] = -1;

	//Generate space for edges
	int vertexCounter = 0;
	//Generate space for vertices
	int edgeCounter = 0;
	int len;
	//Link the graph
	for(vertexCounter=0; vertexCounter<VERTICES; vertexCounter++)
	{
		
		//for gpu transfer

		graph_nodes[vertexCounter].no_of_edges = (rand()%(MAX_EDGES))+1;

		if(vertexCounter ==0)
		{
		graph_nodes[vertexCounter].start= vertexCounter;
		len = graph_nodes[vertexCounter].no_of_edges;
		graph_edge = (int *) malloc(sizeof(int)*len);
				if ((graph_edge = (int *) malloc(sizeof(int)*len))==NULL)
			{
				printf("Could not allocate memory for graph_edge : %d\n", vertexCounter);
				exit(1);
			} 

		}
		else
		{
		graph_nodes[vertexCounter].start= graph_nodes[vertexCounter-1].start+graph_nodes[vertexCounter-1].no_of_edges;
		len += graph_nodes[vertexCounter].no_of_edges;
		graph_edge = (int *) realloc(graph_edge, sizeof(int)*len);
		if ((graph_edge = (int *) realloc(graph_edge, sizeof(int)*len))==NULL)
			{
				printf("Could not reallocate memory for graph_edge: %d\n", vertexCounter);
				exit(1);
			} 
		} 
		
		
		
	//	printf("%d:\t", vertexCounter);
		graph_mask[vertexCounter] = false;
		updating_graph_mask[vertexCounter] = false;
		graph_visited[vertexCounter] = false;
		h_graph_visited[vertexCounter] = false;
		for(edgeCounter=graph_nodes[vertexCounter].start; edgeCounter<(graph_nodes[vertexCounter].no_of_edges+graph_nodes[vertexCounter].start); edgeCounter++)
		{
				int linkedVertex = rand()%VERTICES;
				graph_edge[edgeCounter] = rand()%VERTICES;
				//printf("%d, ", graph_edge[edgeCounter]);
		}

	//	printf("\n");

	}

	// Create the cuda events
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	// Record event on the default stream
	hipEventRecord(start1, 0);

	int presentVertex;
	    for(presentVertex=0;presentVertex<VERTICES;presentVertex++)
	    {
	    	if(!h_graph_visited[presentVertex])
	    	{   		
				Bfs(graph_nodes,graph_edge,presentVertex,h_graph_visited);
				
		}
	    }

	

	// Stop and destroy the timer
	hipEventRecord(stop1,0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&elapsed_gpu1, start1, stop1);
	printf("\nCPU time: %f (msec)\n", elapsed_gpu1);
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
	

	int source=0;

	//set the source node as true in the mask
	graph_mask[source]=true;
	graph_visited[source]=true;

	//Copy the Graph to device memory
	Node *d_graph;
	CUDA_SAFE_CALL(hipMalloc( (void**) &d_graph, sizeof(Node) *VERTICES)) ;
	CUDA_SAFE_CALL(hipMemcpy( d_graph, graph_nodes, sizeof(Node) *VERTICES, hipMemcpyHostToDevice));

	//Copy the Edge List to device Memory
	int* d_edge;
	hipMalloc( (void**) &d_edge, sizeof(int)*(len)) ;
	hipMemcpy( d_edge, graph_edge, sizeof(int)*(len), hipMemcpyHostToDevice) ;


	//Copy the Mask to device memory
	bool* d_graph_mask;
	CUDA_SAFE_CALL(hipMalloc( (void**) &d_graph_mask, sizeof(bool)*VERTICES)) ;
	CUDA_SAFE_CALL(hipMemcpy( d_graph_mask, graph_mask, sizeof(bool)*VERTICES, hipMemcpyHostToDevice));

	bool* d_updating_graph_mask;
	CUDA_SAFE_CALL(hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*VERTICES)) ;
	CUDA_SAFE_CALL(hipMemcpy( d_updating_graph_mask, updating_graph_mask, sizeof(bool)*VERTICES, hipMemcpyHostToDevice)) ;

	//Copy the Visited nodes array to device memory
	bool* d_graph_visited;
	CUDA_SAFE_CALL(hipMalloc( (void**) &d_graph_visited, sizeof(bool)*VERTICES)) ;
	CUDA_SAFE_CALL(hipMemcpy( d_graph_visited, graph_visited, sizeof(bool)*VERTICES, hipMemcpyHostToDevice)) ;


	//make a bool to check if the execution is over
	bool *d_over;
	CUDA_SAFE_CALL(hipMalloc( (void**) &d_over, sizeof(bool)));

	printf("Copied Everything to GPU memory\n");
	
	int num_of_blocks = 1;
	int num_of_threads_per_block = VERTICES;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(VERTICES>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(VERTICES/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}
	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	int k=0;
	printf("Start traversing the tree\n");
	bool over;
	//Call the Kernel untill all the elements of Frontier are not false

	// Create the cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record event on the default stream
	hipEventRecord(start, 0);
	
//	cudaPrintfInit ();
	do
	{
		
		//if no thread changes this value then the loop stops
		over=false;
		hipMemcpy( d_over, &over, sizeof(bool), hipMemcpyHostToDevice) ;
		
		
		Kernel<<< grid, threads, 0 >>>( d_graph,d_edge, d_graph_mask, d_updating_graph_mask, d_graph_visited);
		// check if kernel execution generated and error
		

		Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over);
		// check if kernel execution generated and error
		

		hipMemcpy( &over, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		k++;
	}
	while(over);

	//cudaPrintfDisplay (stdout, true);
//	cudaPrintfEnd ();


	
	
	
	// Stop and destroy the timer
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Kernel Executed %d times\n",k); 
	
	
	
	// cleanup memory
	free( graph_nodes);
	free( graph_edge);
	free( graph_mask);
	free( updating_graph_mask);
	free( graph_visited);
	hipFree(d_graph);
	hipFree(d_edge);
	hipFree(d_graph_mask);
	hipFree(d_updating_graph_mask);
	hipFree(d_graph_visited);

return 0;
}

// Host Implementation
/* crateQueue function takes argument the maximum number of elements the Queue can hold, creates
   a Queue according to it and returns a pointer to the Queue. */
Queue * CreateQueue(int maxElements)
{
        /* Create a Queue */
        Queue *Q;
        Q = (Queue *)malloc(sizeof(Queue));
        /* Initialise its properties */
        Q->elements = (int *)malloc(sizeof(int)*maxElements);
        Q->size = 0;
        Q->capacity = maxElements;
        Q->front = 0;
        Q->rear = -1;
        /* Return the pointer */
        return Q;
}
void Dequeue(Queue *Q)
{
        /* If Queue size is zero then it is empty. So we cannot pop */
        if(Q->size==0)
        {
                printf("Queue is Empty\n");
                return;
        }
        /* Removing an element is equivalent to incrementing index of front by one */
        else
        {
                Q->size--;
                Q->front++;
                /* As we fill elements in circular fashion */
                if(Q->front==Q->capacity)
                {
                        Q->front=0;
                }
        }
        return;
}
int Front(Queue *Q)
{
        if(Q->size==0)
        {
                printf("Queue is Empty\n");
                exit(0);
        }
        /* Return the element which is at the front*/
        return Q->elements[Q->front];
}
void Enqueue(Queue *Q,int element)
{
        /* If the Queue is full, we cannot push an element into it as there is no space for it.*/
        if(Q->size == Q->capacity)
        {
                printf("Queue is Full\n");
        }
        else
        {
                Q->size++;
                Q->rear = Q->rear + 1;
                /* As we fill the queue in circular fashion */
                if(Q->rear == Q->capacity)
                {
                        Q->rear = 0;
                }
                /* Insert the element in its rear side */ 
                Q->elements[Q->rear] = element;
        }
        return;
}



void Bfs(Node* graph_nodes, int* graph_edge, int presentVertex, bool* visited)
{
        visited[presentVertex] = true;
        /* Iterate through all the vertices connected to the presentVertex and perform bfs on those
           vertices if they are not visited before */
        Queue *Q = CreateQueue(VERTICES);
        Enqueue(Q,presentVertex);
        while(Q->size)
        {
                presentVertex = Front(Q);
                printf("Now visiting vertex %d\n",presentVertex);
                Dequeue(Q);
                int iter;
                for(iter=0;iter<graph_nodes[presentVertex].no_of_edges;iter++)
                {
					
                        if(!visited[graph_edge[graph_nodes[presentVertex].start+iter]])
                        {
                                visited[graph_edge[graph_nodes[presentVertex].start+iter]] = 1;
                                Enqueue(Q,graph_edge[graph_nodes[presentVertex].start+iter]);
                        }
					
                }
        }
        return;
        

}

